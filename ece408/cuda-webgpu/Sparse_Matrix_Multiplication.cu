#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLOCK_SIZE 512
/*
   __global__ void SpMV_JDS_T(int num_rows, float *data, 
	int *col_index, int *jds_t_col_ptr, int *jds_row_index, 
	float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
     float dot = 0;
     unsigned in sec = 0;
      while (jds_t_col_ptr[sec+1]-jds_t_col_ptr[sec] > row){
        dot += data[jds_t_col_ptr[sec]+row] * x[col_index[jds_t_col_ptr[sec]+row]];
        sec++;
        }
      y[jds_row_index[row]] = dot;
      }
    }
 */

__global__ void spmvJDSKernel(float *out, int *matColStart, int *matCols,
                              int *matRowPerm, int *matRows,
                              float *matData, float *vec, int dim) {
  //@@ insert spmv kernel for jds format
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < dim) {
    float dot = 0.0;
    int end = matRows[idx];
    for(int sec = 0; sec < end; sec++) {
      dot += matData[matColStart[sec] + idx] * vec[matCols[matColStart[sec] + idx]];
    }
    out[matRowPerm[idx]] = dot;
  }
}

/*
  spmvJDS(
  deviceOutput => out
  deviceJDSColStart => matColStart
  deviceJDSCols => matCols
  deviceJDSRowPerm => matRowPerm
  deviceJDSRows => matRows
  deviceJDSData => matData
  deviceVector => vec 
  dim => dim);
*/
static void spmvJDS(float *out, int *matColStart, int *matCols,
                    int *matRowPerm, int *matRows, float *matData,
                    float *vec, int dim) {

  //@@ invoke spmv kernel for jds format
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid((dim - 1) / BLOCK_SIZE + 1, 1, 1);
  spmvJDSKernel<<<dimGrid, dimBlock>>>(out, matColStart, matCols, matRowPerm, 
                                       matRows, matData, vec, dim);
}

int main(int argc, char **argv) {
  wbArg_t args;
  int *hostCSRCols;
  int *hostCSRRows;
  float *hostCSRData;
  int *hostJDSColStart;
  int *hostJDSCols;
  int *hostJDSRowPerm;
  int *hostJDSRows;
  float *hostJDSData;
  float *hostVector;
  float *hostOutput;
  int *deviceJDSColStart;
  int *deviceJDSCols;
  int *deviceJDSRowPerm;
  int *deviceJDSRows;
  float *deviceJDSData;
  float *deviceVector;
  float *deviceOutput;
  int dim, ncols, nrows, ndata;
  int maxRowNNZ;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostCSRCols = (int *)wbImport(wbArg_getInputFile(args, 0), &ncols, "Integer");
  hostCSRRows = (int *)wbImport(wbArg_getInputFile(args, 1), &nrows, "Integer");
  hostCSRData = (float *)wbImport(wbArg_getInputFile(args, 2), &ndata, "Real");
  hostVector = (float *)wbImport(wbArg_getInputFile(args, 3), &dim, "Real");

  hostOutput = (float *)malloc(sizeof(float) * dim);

  wbTime_stop(Generic, "Importing data and creating memory on host");
  /*
  CSRToJDS(int dim, int *csrRowPtr, int *csrColIdx, float *csrData, int **jdsRowPerm, 
           int **jdsRowNNZ, int **jdsColStartIdx, int **jdsColIdx, float **jdsData);
  */
  CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm, &hostJDSRows,
           &hostJDSColStart, &hostJDSCols, &hostJDSData);
  maxRowNNZ = hostJDSRows[0];

  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
  hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
  hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);

  hipMalloc((void **)&deviceVector, sizeof(float) * dim);
  hipMalloc((void **)&deviceOutput, sizeof(float) * dim);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ,
             hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceVector, hostVector, sizeof(float) * dim, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols, deviceJDSRowPerm, deviceJDSRows,
          deviceJDSData, deviceVector, dim);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceVector);
  hipFree(deviceOutput);
  hipFree(deviceJDSColStart);
  hipFree(deviceJDSCols);
  hipFree(deviceJDSRowPerm);
  hipFree(deviceJDSRows);
  hipFree(deviceJDSData);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, dim);

  free(hostCSRCols);
  free(hostCSRRows);
  free(hostCSRData);
  free(hostVector);
  free(hostOutput);
  free(hostJDSColStart);
  free(hostJDSCols);
  free(hostJDSRowPerm);
  free(hostJDSRows);
  free(hostJDSData);

  return 0;
}

